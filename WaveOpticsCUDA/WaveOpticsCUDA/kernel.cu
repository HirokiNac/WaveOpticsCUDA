#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<stdlib.h>
#include <math.h>

const int threads = 1536;
const int numBlock = 32;
const dim3 threadsPerBlock = dim3(1024);


__global__
void Prop1D_kernel(const double _k,const int _dir,
	const int _n1, const double* _x1, const double* _y1, const double* _u1re, const double* _u1im,
	const int _n2, const double* _x2, const double* _y2, double* _u2re, double* _u2im)
{
	const unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

	double r, rx, ry, rr;
	double tr, ti;
	double tur, tui;
	double ur = 0.0, ui = 0.0;
	if (col < _n2)
	{
		for (int j = 0; j < _n1; j++)
		{
			rx = _x2[col] - _x1[j];
			ry = _y2[col] - _y1[j];
			r = sqrt(rx*rx + ry*ry);

			rr = 1.0 / sqrt(r);
			tr = cos(-_k*r) * rr;
			ti = sin(-_k*r) * rr;

			tur = _u1re[j];
			tui = _u1im[j];

			ur = ur + tur*tr - tui*ti;
			ui = ui + tur*ti + tui*tr;

		}
		_u2re[col] = _u2re[col] + ur;
		_u2im[col] = _u2im[col] + ui;

	}

}

__global__
void Prop2D_kernel(const double _k,const int _dir,
	const int _n1, const double* _x1, const double* _y1, const  double* _z1, const  double* _u1re, const  double* _u1im,
	const int _n2, const double* _x2, const double* _y2, const double* _z2, double* _u2re, double* _u2im)
{
	const unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
	//
	if (_n2 < col)return;

	//
	double r, rx, ry, rz, rr;
	double tr, ti;
	double tur, tui;
	double ur = 0.0, ui = 0.0;
	double x1, y1, z1;

	for (int j = 0; j < _n1; j++)
	{
		x1 = _x1[j];
		y1 = _y1[j];
		z1 = _z1[j];

		rx = _x2[col] - x1;
		ry = _y2[col] - y1;
		rz = _z2[col] - z1;
		r = sqrt(rx*rx + ry*ry + rz*rz);

		rr = 1.0 / r;
		tr = cos(-_k*r) * rr;
		ti = sin(-_k*r) * rr;

		tur = _u1re[j];
		tui = _u1im[j];

		ur = ur + tur*tr - tui*ti;
		ui = ui + tur*ti + tui*tr;

	}
	_u2re[col] = _u2re[col] + ur;
	_u2im[col] = _u2im[col] + ui;
}

extern "C" void
Prop1DCuda(const double _k, const int _dir,
	const int _n1, const double* _x1, const double* _y1, const double* _u1re, const double* _u1im,
	const int _n2, const double* _x2, const double* _y2, double* _u2re, double* _u2im)
{
	hipSetDevice(0);

	size_t memsize1 = _n1 * sizeof(double);
	size_t memsize2 = _n2 * sizeof(double);

	//1
	double *x1 = 0;
	hipMalloc((void**)&x1, memsize1);
	hipMemcpy(x1, _x1, memsize1, hipMemcpyHostToDevice);

	double *y1 = 0;
	hipMalloc((void**)&y1, memsize1);
	hipMemcpy(y1, _y1, memsize1, hipMemcpyHostToDevice);

	double *u1re = 0;
	hipMalloc((void**)&u1re, memsize1);
	hipMemcpy(u1re, _u1re, memsize1, hipMemcpyHostToDevice);

	double *u1im = 0;
	hipMalloc((void**)&u1im, memsize1);
	hipMemcpy(u1im, _u1im, memsize1, hipMemcpyHostToDevice);

	//2
	double *x2 = 0;
	hipMalloc((void**)&x2, memsize2);
	hipMemcpy(x2, _x2, memsize2, hipMemcpyHostToDevice);

	double *y2 = 0;
	hipMalloc((void**)&y2, memsize2);
	hipMemcpy(y2, _y2, memsize2, hipMemcpyHostToDevice);

	double *u2re = 0;
	hipMalloc((void**)&u2re, memsize2);
	//hipMemcpy(u2re, _u2re, memsize2, hipMemcpyHostToDevice);

	double *u2im = 0;
	hipMalloc((void**)&u2im, memsize2);
	//hipMemcpy(u2im, _u2im, memsize2, hipMemcpyHostToDevice);

	Prop1D_kernel << <numBlock, threads >> > (_k, _dir, _n1, x1, y1, u1re, u1im, _n2, x2, y2, u2re, u2im);

	double* u2re_out = 0;
	hipHostMalloc((void**)&u2re_out, memsize2);
	hipMemcpy(u2re_out, u2re, memsize2, hipMemcpyDeviceToHost);
	double* u2im_out = 0;
	hipHostMalloc((void**)&u2im_out, memsize2);
	hipMemcpy(u2im_out, u2im, memsize2, hipMemcpyDeviceToHost);


	for (int i = 0; i < _n2; i++)
	{
		_u2re[i] = u2re_out[i];
		_u2im[i] = u2im_out[i];
	}


	//memfree
	hipFree(x1);
	hipFree(y1);
	hipFree(u1re);
	hipFree(u1im);

	hipFree(x2);
	hipFree(y2);
	hipFree(u2re);
	hipFree(u2im);
	hipFree(u2re_out);
	hipFree(u2im_out);
}

extern "C" void
Prop2DCuda(const double _k,const int _dir,
	const int _n1,const  double* _x1,const  double* _y1,const double* _z1, const double* _u1re,const double* _u1im,
	const int _n2,const double* _x2,const  double* _y2,const double* _z2, double* _u2re, double* _u2im)
{
	hipSetDevice(1);

	size_t memsize1 = _n1 * sizeof(double);
	size_t memsize2 = _n2 * sizeof(double);

	//1
	double *dx1 = 0;
	hipMalloc((void**)&dx1, memsize1);
	hipMemcpy(dx1, _x1, memsize1, hipMemcpyHostToDevice);

	double *dy1 = 0;
	hipMalloc((void**)&dy1, memsize1);
	hipMemcpy(dy1, _y1, memsize1, hipMemcpyHostToDevice);

	double *dz1 = 0;
	hipMalloc((void**)&dz1, memsize1);
	hipMemcpy(dz1, _z1, memsize1, hipMemcpyHostToDevice);

	double *du1re = 0;
	hipMalloc((void**)&du1re, memsize1);
	hipMemcpy(du1re, _u1re, memsize1, hipMemcpyHostToDevice);

	double *du1im = 0;
	hipMalloc((void**)&du1im, memsize1);
	hipMemcpy(du1im, _u1im, memsize1, hipMemcpyHostToDevice);


	//2
	double *dx2 = 0;
	hipMalloc((void**)&dx2, memsize2);
	hipMemcpy(dx2, _x2, memsize2, hipMemcpyHostToDevice);

	double *dy2 = 0;
	hipMalloc((void**)&dy2, memsize2);
	hipMemcpy(dy2, _y2, memsize2, hipMemcpyHostToDevice);

	double *dz2 = 0;
	hipMalloc((void**)&dz2, memsize2);
	hipMemcpy(dz2, _z2, memsize2, hipMemcpyHostToDevice);

	double *du2re = 0;
	hipMalloc((void**)&du2re, memsize2);

	double *du2im = 0;
	hipMalloc((void**)&du2im, memsize2);


	Prop2D_kernel << <numBlock,threads >> >(_k,_dir, _n1, dx1, dy1, dz1, du1re, du1im, _n2, dx2, dy2, dz2, du2re, du2im);

	hipDeviceSynchronize();


	double* u2re_out = (double*)malloc(memsize2);
	//hipMalloc((void**)&u2re_out, memsize2);
	hipMemcpy(u2re_out, du2re, memsize2, hipMemcpyDeviceToHost);
	double* u2im_out = (double*)malloc(memsize2);
	//hipMalloc((void**)&u2im_out, memsize2);
	hipMemcpy(u2im_out, du2im, memsize2, hipMemcpyDeviceToHost);


	for (int i = 0; i < _n2; i++)
	{
		_u2re[i] = u2re_out[i];
		_u2im[i] = u2im_out[i];
	}

	//memfree
	hipFree(dx1);
	hipFree(dy1);
	hipFree(dz1);
	hipFree(du1re);
	hipFree(du1im);

	hipFree(dx2);
	hipFree(dy2);
	hipFree(dz2);
	hipFree(du2re);
	hipFree(du2im);

	free(u2re_out);
	free(u2im_out);
	//hipHostFree(u2re_out);
	//hipHostFree(u2im_out);

	hipDeviceReset();
}


